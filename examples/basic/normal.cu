#include "hip/hip_runtime.h"
#include <vector>
#include <numeric>
#include <iostream>
#include "cuda_mav.cuh"

inline void checkCudaError(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "\rCuda Error " << err << ": " << hipGetErrorString(err) << std::endl;
        std::cerr << "Aborting..." << std::endl;
        exit(1);
    }
}


__global__ void kernel(int prob_size, CudaMav<int> * input, CudaMav<int> * output){
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < prob_size) {
        (*output)[id] = (*input)[id];
    }

}

int main(){
    constexpr int prob_size = 100;
    
    std::vector<int> h_input(prob_size);
    std::iota(h_input.begin(), h_input.end(), 0);
    int * d_input = nullptr;
    checkCudaError(hipMalloc((void**) &d_input, sizeof(int)*prob_size));

    std::vector<int> h_output(prob_size, 0);
    int * d_output = nullptr;
    checkCudaError(hipMalloc((void**) &d_output, sizeof(int)*prob_size));

    checkCudaError(hipMemcpy(d_input, h_input.data(), sizeof(int)* prob_size, hipMemcpyHostToDevice));

    CudaMav<int> input(d_input);
    CudaMav<int> output(d_output);

    constexpr int threads = 32;
    constexpr int blocks = (prob_size/threads)+1;

    kernel<<<blocks, threads>>>(prob_size, input.getDevicePointer(), output.getDevicePointer());
    checkCudaError(hipGetLastError());
    hipDeviceSynchronize();

    auto data = input.getGlobalSettings();

    input.analyze("../../../html/basic_template.html", "../../../out/basic_input.html");
    output.analyze("../../../html/basic_template.html", "../../../out/basic_output.html");

    checkCudaError(hipMemcpy(h_output.data(), d_output, sizeof(int)*prob_size, hipMemcpyDeviceToHost));

    for(auto i = 0; i < h_input.size(); ++i){
        if(h_input[i] != h_output[i]){
            std::cerr << "Element at position " << i << "is not equal (input - output): " << h_input[i] << " != " << h_output[i] << std::endl;
            std::exit(1); 
        }
    }

    checkCudaError(hipFree(d_input));
    checkCudaError(hipFree(d_output));

    std::cout << "kernel finished successful" << std::endl;
    return 0;
}
