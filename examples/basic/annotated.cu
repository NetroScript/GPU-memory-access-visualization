
#include <hip/hip_runtime.h>
#include <vector>
#include <numeric>
#include <iostream>
#include <fstream>

inline void checkCudaError(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "\rCuda Error " << err << ": " << hipGetErrorString(err) << std::endl;
        std::cerr << "Aborting..." << std::endl;
        exit(1);
    }
}

struct MemAccessData {
    int id = 0;
};

__device__ int profile_access(int id, MemAccessData * mem_access){
    mem_access[id].id = id;
    return id;
}

// int * const
// mem_access<int * const>
__global__ void kernel(int prob_size, int * const input, int * output, MemAccessData * mem_access){
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id < prob_size){
        // output[id] = input[id];
        output[id] = input[profile_access(id, mem_access)];
    }
}

// for 1D and 2D: common image format (in best case without extra library)
// or HTML
void visualize(std::vector<MemAccessData> const & mem_accs){
    std::ofstream fs("visu.txt");
    fs << "data\n";
    fs.close();
}

int main(){
    constexpr int prob_size = 100;
    
    std::vector<int> h_input(prob_size);
    std::iota(h_input.begin(), h_input.end(), 0);
    int * d_input = nullptr;
    checkCudaError(hipMalloc((void**) &d_input, sizeof(int)*prob_size));

    std::vector<int> h_output(prob_size, 0);
    int * d_output = nullptr;
    checkCudaError(hipMalloc((void**) &d_output, sizeof(int)*prob_size));

    checkCudaError(hipMemcpy(d_input, h_input.data(), sizeof(int)* prob_size, hipMemcpyHostToDevice));

    std::vector<MemAccessData> h_mem_access(prob_size);
    MemAccessData * d_mem_access = nullptr;
    checkCudaError(hipMalloc((void**) &d_mem_access, sizeof(MemAccessData)*prob_size));

    constexpr int threads = 32;
    constexpr int blocks = (prob_size/threads)+1;

    kernel<<<blocks, threads>>>(prob_size, d_input, d_output, d_mem_access);
    checkCudaError(hipGetLastError());

    checkCudaError(hipMemcpy(h_output.data(), d_output, sizeof(int)*prob_size, hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(h_mem_access.data(), d_mem_access, sizeof(MemAccessData)*prob_size, hipMemcpyDeviceToHost));


    for(auto i = 0; i < h_input.size(); ++i){
        if(h_input[i] != h_output[i]){
            std::cerr << "Element at position " << i << "is not equal (input - output): " << h_input[i] << " != " << h_output[i] << std::endl;
            std::exit(1); 
        }
    }

    visualize(h_mem_access);

    checkCudaError(hipFree(d_input));
    checkCudaError(hipFree(d_output));
    checkCudaError(hipFree(d_mem_access));

    std::cout << "kernel finished successful" << std::endl;
    return 0;
}
